#include "hip/hip_runtime.h"
#include "cudaKernel.h"
#include <helper_functions.h>
#include "hip/hip_runtime.h"
#include ""

#include <cuda_gl_interop.h>

void cudaGetOpenCVImageSize(unsigned int &cols, unsigned int &rows) {
	cols = 640;
	rows = 360;
}

hipGraphicsResource_t vboRes;
hipGraphicsResource_t texRes;

void cudaInit ( unsigned int texId, unsigned int vboId, unsigned int cols, unsigned int rows){

	// Registration with CUDA.
	hipGraphicsGLRegisterImage(&texRes, texId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	hipGraphicsGLRegisterBuffer(&vboRes, vboId, hipGraphicsRegisterFlagsNone);
}

unsigned char *dev_mat_orig = NULL;
unsigned char *dev_histo = NULL;
hipArray* texArray;
float*  vboArray;

__global__ void cudaUncachedHistogramKernel(unsigned char *input, size_t length, int step, float *out) {

	int i = blockIdx.x*blockDim.x*step + threadIdx.x*step;

	if (i < 256) {
		out[3 * i] = 0.0f;
	}
	
	__syncthreads();

	int num_threads = blockDim.x*gridDim.x;

	if (i < length) {
		if (input[i] < 256) {
			//atomicAdd((unsigned int*)&(out[input[i]]), 1);
			atomicAdd((float*)&(out[3*input[i]+1]), 1.0f);
		}
	}

	/*	for (int j = 0; j < 256 * 3; j++) {
		//if (j == 0) printf("sz=%d\n", length);
		out[j] = 0.0f;
	}
	out[4] = 0.1f;*/
}

#define TILE_WIDTH 32
int cudaExecOneStep(unsigned char *data, size_t sz, int step, int channels, int rows, int cols) {
	hipError_t ret;

	if (dev_mat_orig == NULL) {
		ret = hipMalloc((void**)&dev_mat_orig, sz);

		if (ret != hipSuccess) {
			printf("hipMemcpy() error %s\n", hipGetErrorString(ret));
			exit(EXIT_FAILURE);
		}
	}

	/* copy original to device */
	ret = hipMemcpy(dev_mat_orig, data, sz, hipMemcpyHostToDevice);
	if (ret != hipSuccess) {
		printf("hipMemcpy() error %s\n", hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}

	ret = hipGraphicsMapResources(1, &texRes);
	if (ret != hipSuccess) {
		printf("hipMemcpy() error %s\n", hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}
	ret = hipGraphicsMapResources(1, &vboRes);
	if (ret != hipSuccess) {
		printf("hipMemcpy() error %s\n", hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}

	size_t vbosz = 0;

	ret = hipGraphicsSubResourceGetMappedArray(&texArray, texRes, 0, 0);
	if (ret != hipSuccess) {
		printf("hipMemcpy() error %s\n", hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}
	ret = hipGraphicsResourceGetMappedPointer((void**)&vboArray, &vbosz, vboRes);
	if (ret != hipSuccess) {
		printf("hipMemcpy() error %s\n", hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}

	//printf("r*c=%d\n", rows * cols);

	// kernel aufruf hier:
	// dev_mat_orig ist input
	// texArray ist output

	dim3 blocksPerGrid(ceil((input->rows * input->cols * 3) / TILE_WIDTH));
	dim3 threadsPerBlock(TILE_WIDTH);
	cudaUncachedHistogramKernel << <ceil(1.0*(rows * cols) / TILE_WIDTH), TILE_WIDTH >> >(dev_mat_orig, sz, 1, vboArray);

	ret = hipGetLastError();
	if (ret != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	ret = hipDeviceSynchronize();
	if (ret != hipSuccess) {
		fprintf(stderr, "HISTO: hipDeviceSynchronize returned error code %d after launching addKernel! %s\n",
			ret, hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}

	//hipMemset(vboArray, 0, vbosz);

	hipMemcpyToArray(texArray, 0, 0, dev_mat_orig, sz, hipMemcpyDeviceToDevice);

	hipGraphicsUnmapResources(1, &texRes);
	ret = hipGraphicsUnmapResources(1, &vboRes);
	if (ret != hipSuccess) {
		printf("hipMemcpy() error %s\n", hipGetErrorString(ret));
		exit(EXIT_FAILURE);
	}

	return 0;
}
