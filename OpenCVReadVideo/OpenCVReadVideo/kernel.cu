#include "hip/hip_runtime.h"

#include <stdio.h>

// convert the given color image to a grayscale image
// the input data is interpreted as follows:
// the data has number of rows given by "rows"
// the data has number of columns given by "columns"
// each element has size 3bytes each corresponding to one channel of the image b, g, r
// the output data is interpreted as follows:
// the data has number of rows given by "rows"
// the data has number of columns given by "columns"
// each element has size 1byte for one channel (the grayscale channel)
__global__ void colorConvert(unsigned char* grayImage, unsigned char* colorImage, int rows, int columns)
{
	// which pixel does this thread have to work on?
	int column = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
  
	if ((column < columns) && (row < rows)) {
		// calculate offset to access correct element
		int offset = (column)             // offset in a row
					+ (columns * row);    // select row
		// calculate grey values
		unsigned char grayValue = 0.07 * colorImage[offset * 3] + 0.71 * colorImage[offset*3 + 1] + 0.21 * colorImage[offset*3 + 2];
		// copy one value to the result matrix and set other 2 to zero
		// first channel (blue)
		grayImage[offset] = grayValue;
	}
}

__global__ void sobel(unsigned char* outputImage, unsigned char* inputImage, int rows, int columns)
{
	// which pixel does this thread have to work on?
	int column = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	if ((column < columns) && (row < rows)) {
		// calculate offset to access correct element
		int offset = (column)             // offset in a row
			+(columns * row);    // select row

		// the sobel kernels
		int kernelX[] = { 1, 0, -1, 2, 0, -2, 1, 0, -1 };
		int kernelY[] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };		

		// the offsets for the offset
		int offsets[] = {
			- columns - 1,	- columns,	- columns + 1,
			- 1,			0,			1,		
			columns - 1,	columns,	columns + 1		
		};

		// iterate all values in kernelX and 8 neighbours
		unsigned char sobelValueX = 0;
		for (int index = 0; index < 9; index++) {
			int actualOffset = offset + offsets[index];
			if (actualOffset >= 0) {
				sobelValueX += inputImage[actualOffset] * kernelX[index];
			}
		}

		// iterate all values in kernelY and 8 neighbours
		unsigned char sobelValueY = 0;
		for (int index = 0; index < 9; index++) {
			int actualOffset = offset + offsets[index];
			if (actualOffset >= 0) {
				sobelValueY += inputImage[actualOffset] * kernelY[index];
			}
		}

		unsigned char sobelValue = (unsigned char) floor(sqrtf(pow((float)sobelValueX, 2.f) + pow((float)sobelValueY, 2.f))); 
		outputImage[offset] = sobelValue;
	}
}
